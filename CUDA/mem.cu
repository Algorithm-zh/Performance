#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
///opt/cuda/samples/common/inc/hip/hip_runtime_api.h定义了很多函数和宏可以帮我快速检查错误
//找不到可以网上搜一下
//把hip/hip_runtime_api.h和helper_string.h放到根目录里
#include "include/hip/hip_runtime_api.h"

__global__ void kernal(int *pret){
  *pret = 42;
}

int main(){
  //int ret = 0;
  //无论在栈上还是堆上都会报错，因为gpu是独立的显存，不能访问cpu的内存
  //kernal<<<1,1>>>(&ret);
  int *pret;
  //可以用hipMalloc分配显存,同样的cpu也不能访问gpu内存
  checkCudaErrors(hipMalloc(&pret, sizeof(int)));
  kernal<<<1, 1>>>(pret);
  //有了hipMemcpy不需要显示同步
  //checkCudaErrors(hipDeviceSynchronize());

  int ret;
  //通过hipMemcpy可以在cpu和gpu之间拷贝数据
  //并且hipMemcpy会自动进行同步操作，所以不需要进行hipDeviceSynchronize
  checkCudaErrors(hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost));
  printf("result: %d\n", ret);

  hipFree(pret);
  return 0;
}
