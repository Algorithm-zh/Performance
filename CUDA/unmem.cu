#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "include/hip/hip_runtime_api.h"

__global__ void kernal(int *pret){
  *pret = 42;
}


int main(){

  int *pret;
  //um, 统一内存地址，cpu和gpu都可以访问
  //实际上是驱动自动进行的拷贝，省去了自己操作
  //但是会有开销，最好别用
  checkCudaErrors(hipMallocManaged(&pret, sizeof(int)));
  kernal<<<1, 1>>>(pret);
  checkCudaErrors(hipDeviceSynchronize());
  printf("result: %d\n", *pret);
  hipFree(pret);
  return 0;
}
